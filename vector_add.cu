#include <cassert>
#include <iostream>
#include <iomanip>

#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

void check(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int startIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = startIndex; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 24;

    float *A, *B, *C;

    CHECK_CUDA_ERROR(hipHostMalloc(&A, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&B, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&C, N * sizeof(float), hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        A[i] = float(i);
        B[i] = float(i * 2);
    }

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    float *d_a, *d_b, *d_c;
    CHECK_CUDA_ERROR(hipMalloc(&d_a, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, N * sizeof(float)));

    // Time the Host-to-Device memory copy.
    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(stop));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float memcpyHtoD_ms = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&memcpyHtoD_ms, start, stop));
    std::cout << "Time for Host->Device Copy: "
              << std::fixed
              << std::setprecision(3)
              << memcpyHtoD_ms
              << " ms"
              << std::endl;

    int blockSize = 256;
    hipDeviceProp_t props;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&props, 0));
    std::cout << "Number of Streaming Multiprocessors (SMs): "
              << props.multiProcessorCount << std::endl;
    int gridSize = props.multiProcessorCount * 5; // (N + blockSize - 1) / blockSize;

    // Time the Kernel Execution.
    CHECK_CUDA_ERROR(hipEventRecord(start));
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
    CHECK_CUDA_ERROR(hipEventRecord(stop));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float kernel_ms = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&kernel_ms, start, stop));
    std::cout << "Time for Kernel Execution: "
              << std::fixed
              << std::setprecision(3)
              << kernel_ms
              << " ms"
              << std::endl;

    // Time the Device-to-Host memory copy.
    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(stop));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float memcpyDtoH_ms = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&memcpyDtoH_ms, start, stop));
    std::cout << "Time for Device->Host Copy: "
              << std::fixed
              << std::setprecision(3)
              << memcpyDtoH_ms
              << " ms"
              << std::endl;

    std::cout << "Verifying results..." << std::endl;
    bool success = true;
    for (int i = 0; i < 10; ++i) { // Check first 10.
        if (std::abs(C[i] - (A[i] + B[i])) > 1e-5) {
            std::cerr << "Verification failed at index " << i << std::endl;
            success = false;
            break;
        }
    }
    if (success) {
        for (int i = N - 10; i < N; ++i) { // Check last 10.
            if (std::abs(C[i] - (A[i] + B[i])) > 1e-5) {
                std::cerr << "Verification failed at index " << i << std::endl;
                success = false;
                break;
            }
        }
    }

    if (success) {
        std::cout << "Verification successful!" << std::endl;
    }

    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));
    CHECK_CUDA_ERROR(hipHostFree(A));
    CHECK_CUDA_ERROR(hipHostFree(B));
    CHECK_CUDA_ERROR(hipHostFree(C));

    return 0;
}

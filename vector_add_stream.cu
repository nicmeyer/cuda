#include <cassert>
#include <iostream>
#include <iomanip>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

void check(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int startIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = startIndex; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 24;

    float *A, *B, *C;

    CHECK_CUDA_ERROR(hipHostMalloc(&A, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&B, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&C, N * sizeof(float), hipHostMallocDefault));

    for (int i = 0; i < N; ++i) {
        A[i] = float(i);
        B[i] = float(i * 2);
    }

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    float *d_a, *d_b, *d_c;
    CHECK_CUDA_ERROR(hipMalloc(&d_a, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, N * sizeof(float)));

    const int nStreams = 4;
    std::vector<hipStream_t> streams(nStreams);
    for (int i = 0; i < nStreams; ++i) {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
    }

    int chunkSize = (N + nStreams - 1) / nStreams;

    int blockSize = 256;
    hipDeviceProp_t props;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&props, 0));
    int gridSize = props.multiProcessorCount * 5;

    std::cout << "Starting pipelined execution with "
              << nStreams
              << " streams..."
              << std::endl;

    // Time the entire pipelined operation.
    CHECK_CUDA_ERROR(hipEventRecord(start));

    for (int i = 0; i < nStreams; ++i) {
        int offset = i * chunkSize;
        int currentSize = (i == nStreams - 1) ? (N - offset) : chunkSize;
        if (currentSize <= 0) continue;

        CHECK_CUDA_ERROR(hipMemcpyAsync(d_a + offset, A + offset, currentSize * sizeof(float),
            hipMemcpyHostToDevice, streams[i]));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_b + offset, B + offset, currentSize * sizeof(float),
            hipMemcpyHostToDevice, streams[i]));

        vectorAdd<<<gridSize, blockSize, 0, streams[i]>>>(d_a + offset, d_b + offset, d_c + offset, currentSize);

        CHECK_CUDA_ERROR(hipMemcpyAsync(C + offset, d_c + offset, currentSize * sizeof(float),
            hipMemcpyDeviceToHost, streams[i]));
    }

    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    float total_ms = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&total_ms, start, stop));
    std::cout << "Total Time for Pipelined Execution: "
              << std::fixed << std::setprecision(3) << total_ms << " ms" << std::endl;

    for (int i = 0; i < nStreams; ++i) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }

    std::cout << "Verifying results..." << std::endl;
    bool success = true;
    for (int i = 0; i < N; i += N / 10) { // Check first 10.
        if (std::abs(C[i] - (A[i] + B[i])) > 1e-5) {
            std::cerr << "Verification failed at index " << i << std::endl;
            success = false;
            break;
        }
    }
    if (success) {
        for (int i = N - 10; i < N; ++i) { // Check last 10.
            if (std::abs(C[i] - (A[i] + B[i])) > 1e-5) {
                std::cerr << "Verification failed at index " << i << std::endl;
                success = false;
                break;
            }
        }
    }
    if (success) {
        std::cout << "Verification successful!" << std::endl;
    }

    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));
    CHECK_CUDA_ERROR(hipHostFree(A));
    CHECK_CUDA_ERROR(hipHostFree(B));
    CHECK_CUDA_ERROR(hipHostFree(C));

    return 0;
}
